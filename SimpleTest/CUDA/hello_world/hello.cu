#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    // Launch the kernel with a single thread
    cuda_hello<<<1,1>>>();

    // Check for any errors during kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for the GPU to finish before accessing the result
    hipDeviceSynchronize();

    return 0;
}
