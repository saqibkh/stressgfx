
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);
    // Allocate device memory for b
    hipMalloc((void**)&d_b, sizeof(float) * N);
    // Allocate device memory for out
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
    hipFree(d_b);
    free(b);
    hipFree(d_out);
    free(out);
}
